#include "hip/hip_runtime.h"
#include "Vec3.cuh"

__host__ __device__ float RayTracer::Vec3::magnitude() const {
    return sqrtf(x*x + y*y + z*z);
}

__host__ __device__ RayTracer::Vec3 &RayTracer::Vec3::operator-() {
    x = -x;
    y = -y;
    z = -z;
    return *this;
}

__host__ __device__ RayTracer::Vec3 &RayTracer::Vec3::operator+=(const RayTracer::Vec3 &v) {
    x += v.x;
    y += v.y;
    z += v.z;
    return *this;
}

__host__ __device__ RayTracer::Vec3 &RayTracer::Vec3::operator-=(const RayTracer::Vec3 &v) {
    x -= v.x;
    y -= v.y;
    z -= v.z;
    return *this;
}

__host__ __device__ RayTracer::Vec3 &RayTracer::Vec3::operator*=(const RayTracer::Vec3 &v) {
    x *= v.x;
    y *= v.y;
    z *= v.z;
    return *this;
}

__host__ __device__ RayTracer::Vec3 &RayTracer::Vec3::operator/=(const RayTracer::Vec3 &v) {
    x /= v.x;
    y /= v.y;
    z /= v.z;
    return *this;
}

__host__ __device__ RayTracer::Vec3 &RayTracer::Vec3::operator+=(float n) {
    x += n;
    y += n;
    z += n;
    return *this;
}

__host__ __device__ RayTracer::Vec3 &RayTracer::Vec3::operator-=(float n) {
    x -= n;
    y -= n;
    z -= n;
    return *this;
}

__host__ __device__ RayTracer::Vec3 &RayTracer::Vec3::operator*=(float n) {
    x *= n;
    y *= n;
    z *= n;
    return *this;
}

__host__ __device__ RayTracer::Vec3 &RayTracer::Vec3::operator/=(float n) {
    x /= n;
    y /= n;
    z /= n;
    return *this;
}

__host__ __device__ RayTracer::Vec3 RayTracer::normalize(const RayTracer::Vec3 &v) {
    const float mag = v.magnitude();
    return {v.x / mag, v.y / mag, v.z / mag};
}
